#define _USE_MATH_DEFINES
#define FMT_HEADER_ONLY

#include "fmt/format.h"
#include "fmt/ranges.h"

#include "hip/hip_runtime.h"
#include "hipblas.h"

#include <array>
#include <cmath>
#include <cstdlib>

constexpr int blas_index(int row, int column, int number_rows) {
    return column * number_rows + row;
}

static __inline__ void modify (
    hipblasHandle_t handle,
    float *matrix,
    int number_rows,
    int number_columns,
    int row,
    int column,
    float alpha,
    float beta
) {
    /*!
        `hipblasSscal` Function to scale a vector by a given scalar (in-place).
    
        @param[in] handle:
            Handle of the cuBLAS instance.
        @param[in] n:
            Size of the vector.
        @param[in] alpha:
            Pointer to the constant to scale the vector with.
        @param[in,out] x:
            Pointer to the first element of the vector.
        @param[in] incx:
            Stride between consecutive elements of the vector.
        @return:
            Status of the operation (handler).
    */
    // Scale the elements in the given row after the given column by alpha.
    int size_alpha { number_columns - column };
    hipblasSscal(
        handle,
        size_alpha,
        &alpha,
        &matrix[blas_index(row, column, number_rows)],
        number_rows
    );
    // Scale the elements in the given column after the given row by beta.
    int size_beta { number_rows - row };
    hipblasSscal(
        handle,
        size_beta,
        &beta,
        &matrix[blas_index(row, column, number_rows)],
        1
    );
}

int main (void){
    constexpr int number_rows { 6 };
    constexpr int number_columns { 5 };
    hipError_t cuda_status;
    hipblasStatus_t blas_status;
    hipblasHandle_t handle;
    std::array<float, number_rows * number_columns> matrix;
    float * device_matrix;

    fmt::print("\n...input:\n");
    for (int row { 0 }; row < number_rows; ++row) {
        for (int column { 0 }; column < number_columns; ++column) {
            matrix[blas_index(row, column, number_rows)] =
                static_cast<float>(row * number_columns + column + 1);
            fmt::print("{:10.2f}", matrix[blas_index(row, column, number_rows)]);
        }
        fmt::print("\n");
    }

    cuda_status = hipMalloc(&device_matrix, number_rows * number_columns * sizeof(float));
    if (cuda_status != hipSuccess) {
        fmt::print("...device memory allocation failed!\n");
        return EXIT_FAILURE;
    }

    blas_status = hipblasCreate(&handle);
    if (blas_status != HIPBLAS_STATUS_SUCCESS) {
        fmt::print("...cuBLAS initialization failed!\n");
        return EXIT_FAILURE;
    }
    blas_status = hipblasSetMatrix(
        number_rows,
        number_columns,
        sizeof(float),
        matrix.data(),
        number_rows,
        device_matrix,
        number_rows
    );
    if (blas_status != HIPBLAS_STATUS_SUCCESS) {
        fmt::print("...data download failed!\n");
        hipFree(device_matrix);
        hipblasDestroy(handle);
        return EXIT_FAILURE;
    }

    modify(handle, device_matrix, number_rows, number_columns, 1, 2, 16.0f, 12.0f);

    blas_status = hipblasGetMatrix(
        number_rows,
        number_columns,
        sizeof(float),
        device_matrix,
        number_rows,
        matrix.data(),
        number_rows
    );
    if (blas_status != HIPBLAS_STATUS_SUCCESS) {
        fmt::print("...data upload failed!\n");
        hipFree(device_matrix);
        hipblasDestroy(handle);
        return EXIT_FAILURE;
    }

    hipFree(device_matrix);
    hipblasDestroy(handle);

    fmt::print("\n...output:\n");
    for (int row { 0 }; row < number_rows; ++row) {
        for (int column { 0 }; column < number_columns; ++column) {
            fmt::print("{:10.2f}", matrix[blas_index(row, column, number_rows)]);
        }
        fmt::print("\n");
    }
    
    return EXIT_SUCCESS;
}