#include "hip/hip_runtime.h"
#include "dgemm.h"

#include <omp.h>

#include <iostream>
#include <string>
#include <stdexcept>

__global__ void do_dgemm(double* A, double* B, double* C, int rows, int middle, int columns)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < rows && j < columns)
        for (int k = 0; k < middle; k++)
            C[i * columns + j] += A[i * middle + k] * B[k * columns + j];
}

void cuda_dgemm(const std::vector<double>& A, const std::vector<double>& B, std::vector<double>& C, int rows, int middle, int columns)
{
    double *d_A, *d_B, *d_C;
    hipMalloc(&d_A, sizeof(double) * rows * middle);
    hipMalloc(&d_B, sizeof(double) * middle * columns);
    hipMalloc(&d_C, sizeof(double) * rows * columns);

    hipMemcpy(d_A, A.data(), sizeof(double) * rows * middle, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B.data(), sizeof(double) * middle * columns, hipMemcpyHostToDevice);
    hipMemcpy(d_C, C.data(), sizeof(double) * rows * columns, hipMemcpyHostToDevice);

    dim3 block(32, 32);
    dim3 grid(std::ceil(rows / 32.0), std::ceil(columns / 32.0));

    do_dgemm<<<grid, block>>>(d_A, d_B, d_C, rows, middle, columns);

    hipMemcpy(C.data(), d_C, sizeof(double) * rows * columns, hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}

void check(const std::vector<double>& C, int middle)
{
    for (double value : C)
        if (value != middle * 2)
            throw std::runtime_error("Incorrect result: " + std::to_string(value));
}