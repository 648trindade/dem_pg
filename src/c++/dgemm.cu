#include "hip/hip_runtime.h"
#include "dgemm.h"

#include <omp.h>

#include <string>
#include <stdexcept>

__global__ void do_dgemm()
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    printf("Hello World from %d!\n", index);
}

void cuda_dgemm(const std::vector<double>& A, const std::vector<double>& B, std::vector<double>& C, int rows, int middle, int columns)
{
    double *d_A, *d_B, *d_C;
    hipMalloc(&d_A, sizeof(double) * rows * middle);
    hipMalloc(&d_B, sizeof(double) * middle * columns);
    hipMalloc(&d_C, sizeof(double) * rows * columns);

    hipMemcpy(d_A, A.data(), sizeof(double) * rows * middle, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B.data(), sizeof(double) * middle * columns, hipMemcpyHostToDevice);
    hipMemcpy(d_C, C.data(), sizeof(double) * rows * columns, hipMemcpyHostToDevice);

    //dim3 block();
    do_dgemm<<<2, 4>>>();

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}

void check(const std::vector<double>& C, int middle)
{
    for (double value : C)
        if (value != middle * 2)
            throw std::runtime_error("Incorrect result: " + std::to_string(value));
}